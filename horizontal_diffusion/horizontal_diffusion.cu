#include "hip/hip_runtime.h"
#include "horizontal_diffusion.h"
#include "../repository.hpp"
#include "../utils.hpp"
#include "horizontal_diffusion_reference.hpp"
#include "../functions.hpp"

#define BLOCK_X_SIZE 28
#define BLOCK_Y_SIZE 8

#define HALO_BLOCK_X_MINUS 1
#define HALO_BLOCK_X_PLUS 1

#define HALO_BLOCK_Y_MINUS 1
#define HALO_BLOCK_Y_PLUS 1

#define PADDED_BOUNDARY 1

// #define __ldg( a ) a
#define REF &
// #define REF

inline __device__ unsigned int cache_index(const unsigned int ipos, const unsigned int jpos) {
    return (ipos) +
        (jpos) * ( BLOCK_X_SIZE
                   + HALO_BLOCK_X_MINUS + HALO_BLOCK_X_PLUS);
}

inline __device__ unsigned int cache_index_in(const unsigned int ipos, const unsigned int jpos) {
    return (ipos) +
        (jpos) * ( BLOCK_X_SIZE
                   + HALO_BLOCK_X_MINUS + HALO_BLOCK_X_PLUS +2 );
}

__global__ void cukernel(
    Real *in, Real *out, Real *coeff, const IJKSize domain, const IJKSize halo, const IJKSize strides) {

    int ipos, jpos;

    // set the thread position by default out of the block
    ipos = blockIdx.x * BLOCK_X_SIZE + threadIdx.x;
    jpos = blockIdx.y * BLOCK_Y_SIZE + threadIdx.y;

    int index_ = index(ipos, jpos, 0, strides);
// flx and fly can be defined with smaller cache sizes, however in order to reuse the same cache_index function, I
// defined them here
// with same size. shared memory pressure should not be too high nevertheless
#define CACHE_SIZE ( BLOCK_X_SIZE + HALO_BLOCK_X_MINUS + HALO_BLOCK_X_PLUS) * (BLOCK_Y_SIZE + 2)
#define CACHE_SIZE_IN ( BLOCK_X_SIZE+2 + HALO_BLOCK_X_MINUS + HALO_BLOCK_X_PLUS) * (BLOCK_Y_SIZE+4)
    __shared__ Real in_s[CACHE_SIZE_IN];
    __shared__ Real lap[CACHE_SIZE];
    __shared__ Real flx[CACHE_SIZE];
    __shared__ Real fly[CACHE_SIZE];
    Real in_ip1,
        in_jp1,
        in_reg_,
        in_center,
        lap_center;
    // if (ipos<domain.m_i && jpos<domain.m_j && threadIdx.x>1 && threadIdx.x<BLOCK_X_SIZE+2 && threadIdx.y>1 && threadIdx.y<BLOCK_Y_SIZE+2 ) {
    //     coeff_rp1 = __ldg(REF coeff[index_]);
    // }

    if( ipos < domain.m_i && jpos < domain.m_j ){
        // in_s[cache_index_in(threadIdx.x, threadIdx.y)] = __ldg(REF in[index_]);
        // __syncthreads();
        // in_s[cache_index_in(threadIdx.x, threadIdx.y)] = __ldg(REF in[index_]);
        // __syncthreads();

        for (int kpos = 0; kpos < domain.m_k; ++kpos) {
            // if ( ipos<domain.m_i && jpos<domain.m_j && threadIdx.x>1 && threadIdx.x<30 && threadIdx.y>1 && threadIdx.y<BLOCK_Y_SIZE+2 ) {
            //     coeff_r = coeff_rp1;
            //     if( kpos<domain.m_k-1 )
            //         coeff_rp1 = __ldg(REF coeff[index_+index(0,0,1,strides)]);
            // }

            // if (is_in_domain< -2, 2, -2, 2 >(iblock_pos, jblock_pos, block_size_i, block_size_j)) {

            // }
            in_s[cache_index_in(threadIdx.x, threadIdx.y)] = __ldg(REF in[index_]);
            __syncthreads();
            // in_im1 = in_s[cache_index_in(threadIdx.x-1, threadIdx.y)];
            // if(threadIdx.y>0)
            // in_jm1 = in_s[cache_index_in(threadIdx.x, threadIdx.y-1)];

            // in_s[cache_index_in(threadIdx.x, threadIdx.y)] = __ldg(REF in[index_]); //in_reg_;
            // __syncthreads();

            if(kpos < domain.m_k-1)
                in_reg_ = __ldg(REF in[index_ + index(0, 0, 1, strides)]);

            if (ipos<domain.m_i && jpos<domain.m_j && threadIdx.x>0 && threadIdx.x<BLOCK_X_SIZE+3 && threadIdx.y>0 && threadIdx.y<BLOCK_Y_SIZE+3) {
                in_center = in_s[cache_index_in(threadIdx.x, threadIdx.y)];
                in_ip1 = in_s[cache_index_in(threadIdx.x+1, threadIdx.y)];
                in_jp1 = in_s[cache_index_in(threadIdx.x, threadIdx.y+1)];
                lap[cache_index(threadIdx.x, threadIdx.y)] =
                    (Real)4 * __ldg( REF in[index_] ) -
                    ( in_ip1 + in_s[cache_index_in(threadIdx.x-1, threadIdx.y)] +
                      in_jp1 + in_s[cache_index_in(threadIdx.x, threadIdx.y-1)]);
                lap_center = lap[cache_index(threadIdx.x, threadIdx.y)];
            }

            __syncthreads();

            if (//is_in_domain< -1, 0, 0, 0 >(iblock_pos, jblock_pos, block_size_i, block_size_j)
                ipos<domain.m_i && jpos<domain.m_j && threadIdx.x>0 && threadIdx.x<BLOCK_X_SIZE+2 && threadIdx.y>0+1 && threadIdx.y<BLOCK_Y_SIZE+3-1 ) {
                flx[cache_index(threadIdx.x, threadIdx.y)] =
                    lap[cache_index(threadIdx.x+1, threadIdx.y)] - lap_center;
                if (flx[cache_index(threadIdx.x, threadIdx.y)] *
                    (in_ip1 - in_center) >
                    0) {
                    flx[cache_index(threadIdx.x, threadIdx.y)] = 0.;
                }
            }

            if (//is_in_domain< 0, 0, -1, 0 >(iblock_pos, jblock_pos, block_size_i, block_size_j)
                ipos<domain.m_i && jpos<domain.m_j && threadIdx.x>0*+1 && threadIdx.x<BLOCK_X_SIZE+3-1 && threadIdx.y>0 && threadIdx.y<BLOCK_Y_SIZE+2) {
                fly[cache_index(threadIdx.x, threadIdx.y)] =
                    lap[cache_index(threadIdx.x, threadIdx.y + 1)] - lap_center;
                if (fly[cache_index(threadIdx.x, threadIdx.y)] *
                    (in_jp1 - in_center) >
                    0) {
                    fly[cache_index(threadIdx.x, threadIdx.y)] = 0.;
                }
            }

            __syncthreads();

            if (// is_in_domain< 0, 0, 0, 0 >(iblock_pos, jblock_pos, block_size_i, block_size_j)
                ipos<domain.m_i && jpos<domain.m_j && threadIdx.x>1 && threadIdx.x<BLOCK_X_SIZE+2 && threadIdx.y>1 && threadIdx.y<BLOCK_Y_SIZE+2 ) {
                // printf("i: %d, j: %d => %d \n", ipos, jpos, index_);
                out[index_] =
                    in_center -
                    __ldg(REF coeff[index_])
                    *
                    ( flx[cache_index(threadIdx.x, threadIdx.y)] - flx[cache_index(threadIdx.x-1, threadIdx.y)] +
                     fly[cache_index(threadIdx.x, threadIdx.y)] - fly[cache_index(threadIdx.x, threadIdx.y - 1)]);
            }
            in_s[cache_index_in(threadIdx.x, threadIdx.y)] = in_reg_;
            // if( ipos < domain.m_i && jpos < domain.m_j )
            // {
            //     printf("ipos %d, jpos %d, index_ %d, cache_index %d \n", ipos, jpos, index_, cache_index_in(threadIdx.x, threadIdx.y));
            //     out[index_] = in_s[cache_index_in(iblock_pos, jblock_pos)];
            // }

            index_ += index(0,0,1, strides);

        }
    }
}

void launch_kernel(repository &repo, timer_cuda* time) {
    IJKSize domain = repo.domain();
    IJKSize halo = repo.halo();

    dim3 threads, blocks;
    threads.x = BLOCK_X_SIZE +4;
    threads.y = BLOCK_Y_SIZE+4;//BLOCK_Y_SIZE + HALO_BLOCK_Y_MINUS + HALO_BLOCK_Y_PLUS + (HALO_BLOCK_X_MINUS > 0 ? 1 : 0) +
    //(HALO_BLOCK_X_PLUS > 0 ? 1 : 0);
    threads.z = 1;
    blocks.x = (domain.m_i + BLOCK_X_SIZE - 1) /  BLOCK_X_SIZE;
    blocks.y = (domain.m_j + BLOCK_Y_SIZE - 1) / BLOCK_Y_SIZE;
    blocks.z = 1;

    IJKSize strides;
    compute_strides(domain, halo, strides);

    Real *in = repo.field_d("u_in");
    Real *out = repo.field_d("u_out");
    Real *coeff = repo.field_d("coeff");

    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(cukernel), hipSharedMemBankSizeEightByte);

    if(time) time->start();
    cukernel<<< blocks, threads, 0 >>>(in, out, coeff, domain, halo, strides);
    if(time) time->pause();
}
